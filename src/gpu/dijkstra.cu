#include "hip/hip_runtime.h"
#include "src/dijkstra.hpp"

#include "Utilities.cuh"

#define NUM_ASYNCHRONOUS_ITERATIONS 20  // Number of async loop iterations before attempting to read results back

#define BLOCK_SIZE 16

bool allFinalizedVertices(std::vector<bool> &finalizedVertices, int numVertices) {

    for (int i = 0; i < numVertices; i++)  if (finalizedVertices[i] == true) { return false; }

    return true;
}

__global__ void initializeArrays(bool * __restrict__ d_finalizedVertices, float* __restrict__ d_shortestDistances, float* __restrict__ d_updatingShortestDistances,
                                 const int sourceVertex, const int numVertices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (sourceVertex == tid) {

            d_finalizedVertices[tid]            = true;
            d_shortestDistances[tid]            = 0.f;
            d_updatingShortestDistances[tid]    = 0.f; }

        else {

            d_finalizedVertices[tid]            = false;
            d_shortestDistances[tid]            = FLT_MAX;
            d_updatingShortestDistances[tid]    = FLT_MAX;
        }
    }
}

__global__  void Kernel1(const int * __restrict__ vertexArray, const int* __restrict__ edgeArray,
                         const float * __restrict__ weightArray, bool * __restrict__ finalizedVertices, float* __restrict__ shortestDistances,
                         float * __restrict__ updatingShortestDistances, const int numVertices, const int numEdges) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (finalizedVertices[tid] == true) {

            finalizedVertices[tid] = false;

            int edgeStart = vertexArray[tid], edgeEnd;

            if (tid + 1 < (numVertices)) edgeEnd = vertexArray[tid + 1];
            else                         edgeEnd = numEdges;

            for (int edge = edgeStart; edge < edgeEnd; edge++) {
                int nid = edgeArray[edge];
                atomicMin(&updatingShortestDistances[nid], shortestDistances[tid] + weightArray[edge]);
            }
        }
    }
}

__global__  void Kernel2(const int * __restrict__ vertexArray, const int * __restrict__ edgeArray, const float* __restrict__ weightArray,
                         bool * __restrict__ finalizedVertices, float* __restrict__ shortestDistances, float* __restrict__ updatingShortestDistances,
                         const int numVertices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (shortestDistances[tid] > updatingShortestDistances[tid]) {
            shortestDistances[tid] = updatingShortestDistances[tid];
            finalizedVertices[tid] = true; }

        updatingShortestDistances[tid] = shortestDistances[tid];
    }
}

std::vector<float> dijkstra_cuda(const Graph &graph, int sourceVertex)
{

    // --- Create device-side adjacency-list, namely, vertex array Va, edge array Ea and weight array Wa from G(V,E,W)
    int     *d_vertexArray;         gpuErrchk(hipMalloc(&d_vertexArray, sizeof(int)   * graph.vertex_array.size()));
    int     *d_edgeArray;           gpuErrchk(hipMalloc(&d_edgeArray,   sizeof(int)   * graph.edge_array.size()));
    float   *d_weightArray;         gpuErrchk(hipMalloc(&d_weightArray, sizeof(float) * graph.weight_array.size()));

    // --- Copy adjacency-list to the device
    gpuErrchk(hipMemcpy(d_vertexArray, graph.vertex_array.data(), sizeof(int) * graph.vertex_array.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_edgeArray,   graph.edge_array.data(), sizeof(int) * graph.edge_array.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_weightArray, graph.weight_array.data(), sizeof(float) * graph.weight_array.size(), hipMemcpyHostToDevice));

    // --- Create mask array Ma, cost array Ca and updating cost array Ua of size V
    bool    *d_finalizedVertices;           gpuErrchk(hipMalloc(&d_finalizedVertices, sizeof(bool)   * graph.vertex_array.size()));
    float   *d_shortestDistances;           gpuErrchk(hipMalloc(&d_shortestDistances, sizeof(float) * graph.vertex_array.size()));
    float   *d_updatingShortestDistances;   gpuErrchk(hipMalloc(&d_updatingShortestDistances, sizeof(float) * graph.vertex_array.size()));

    bool * h_finalizedVertices = new bool[graph.vertex_array.size()];

    // --- Initialize mask Ma to false, cost array Ca and Updating cost array Ua to \u221e
    initializeArrays <<<iDivUp(graph.vertex_array.size(), BLOCK_SIZE), BLOCK_SIZE >>>(d_finalizedVertices, d_shortestDistances,
                               d_updatingShortestDistances, sourceVertex, graph.vertex_array.size());
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // --- Read mask array from device -> host
    gpuErrchk(hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * graph.vertex_array.size(), hipMemcpyDeviceToHost));

    while (std::all_of(h_finalizedVertices, h_finalizedVertices + graph.vertex_array.size(), [](bool x) { return !x; })) {

        // --- In order to improve performance, we run some number of iterations without reading the results.  This might result
        //     in running more iterations than necessary at times, but it will in most cases be faster because we are doing less
        //     stalling of the GPU waiting for results.
        for (int asyncIter = 0; asyncIter < NUM_ASYNCHRONOUS_ITERATIONS; asyncIter++) {

            Kernel1 <<<iDivUp(graph.vertex_array.size(), BLOCK_SIZE), BLOCK_SIZE >>>(d_vertexArray, d_edgeArray, d_weightArray, d_finalizedVertices, d_shortestDistances,
                                                            d_updatingShortestDistances, graph.vertex_array.size(), graph.edge_array.size());
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
            Kernel2 <<<iDivUp(graph.vertex_array.size(), BLOCK_SIZE), BLOCK_SIZE >>>(d_vertexArray, d_edgeArray, d_weightArray, d_finalizedVertices, d_shortestDistances, d_updatingShortestDistances,
                                                            graph.vertex_array.size());
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        gpuErrchk(hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * graph.vertex_array.size(), hipMemcpyDeviceToHost));
    }

    // --- Copy the result to host
    std::vector<float> shortest_distance(graph.vertex_array.size());
    gpuErrchk(hipMemcpy(shortest_distance.data(), d_shortestDistances, sizeof(float) * shortest_distance.size(), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_vertexArray));
    gpuErrchk(hipFree(d_edgeArray));
    gpuErrchk(hipFree(d_weightArray));
    gpuErrchk(hipFree(d_finalizedVertices));
    gpuErrchk(hipFree(d_shortestDistances));
    gpuErrchk(hipFree(d_updatingShortestDistances));

    return shortest_distance;
}